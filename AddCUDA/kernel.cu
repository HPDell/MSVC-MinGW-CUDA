#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "kernel.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
bool addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }

    // Launch a kernel on the GPU with one thread for each element.
	dim3 blockSize(256), gridSize((size + blockSize.x - 1) / blockSize.x);
    addKernel<<<gridSize, blockSize >>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		return false;
    }
    
    return true;
}
